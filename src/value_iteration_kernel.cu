#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>

#include <cstdint>

#include "value_iteration2/ValueIterator.h"

using namespace value_iteration2;

__device__ int toIndex(int ix, int iy, int it, int cell_num_x, int cell_num_t) {
  return it + ix * cell_num_t + iy * (cell_num_t * cell_num_x);
}

__device__ uint64_t actionCost(State& s, Action& a, State* states,
                               uint64_t max_cost, int num_states,
                               int cell_num_x, int cell_num_y, int cell_num_t,
                               uint64_t prob_base_bit) {
  uint64_t cost = 0;
  for (auto& tran : a._state_transitions[s.it_]) {
    int ix = s.ix_ + tran._dix;
    if (ix < 0 || ix >= cell_num_x) return max_cost;

    int iy = s.iy_ + tran._diy;
    if (iy < 0 || iy >= cell_num_y) return max_cost;

    int it = (tran._dit + cell_num_t) % cell_num_t;

    auto& after_s = states[toIndex(ix, iy, it, cell_num_x, cell_num_t)];
    if (!after_s.free_) return max_cost;

    cost += (after_s.total_cost_ + after_s.penalty_ + after_s.local_penalty_) *
            tran._prob;
  }

  return cost >> prob_base_bit;
}

__global__ void valueIterationKernel(State* states, Action* actions,
                                     int num_states, int num_actions,
                                     uint64_t max_cost, int cell_num_x,
                                     int cell_num_y, int cell_num_t,
                                     uint64_t prob_base_bit) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < num_states) {
    State& s = states[idx];
    if (!s.free_ || s.final_state_) return;

    uint64_t min_cost = max_cost;
    Action* min_action = nullptr;

    for (int a_idx = 0; a_idx < num_actions; ++a_idx) {
      Action& a = actions[a_idx];
      uint64_t cost = actionCost(s, a, states, max_cost, num_states, cell_num_x,
                                 cell_num_y, cell_num_t, prob_base_bit);

      if (cost < min_cost) {
        min_cost = cost;
        min_action = &a;
      }
    }

    s.total_cost_ = min_cost;
    s.optimal_action_ = min_action;
  }
}

__global__ void setStateKernel(State* states, nav_msgs::msg::OccupancyGrid map,
                               double safety_radius, int cell_num_x,
                               int cell_num_y, int cell_num_t) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;

  if (idx < cell_num_x * cell_num_y * cell_num_t) {
    int ix = idx % cell_num_x;
    int iy = (idx / cell_num_x) % cell_num_y;
    int it = idx / (cell_num_x * cell_num_y);

    unsigned int cost = (unsigned int)(map.data[ix + cell_num_x * iy] & 0xFF);
    states[idx] = State(ix, iy, it, cost);
  }
}
